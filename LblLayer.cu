#include "hip/hip_runtime.h"
#include "LblLayer.cuh"
#include "DataLayer.h"

__global__ void getDiffData(float* targets, float* diffData, int label_count, int _batch_size) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= _batch_size)
		return;
	const int label_value = static_cast<int>(targets[idx]);
	diffData[idx * label_count + label_value] -= 1;
}

void LblLayer::copytoDevData(float* imagesfloat, int imageID) {
	int inputsize = inchannel*inheight*inwidth;
	error->checkError(hipMemcpyAsync(ptrToOutData, &imagesfloat[imageID *  inputsize], sizeof(float) * inputsize, hipMemcpyHostToDevice));
	error->checkError(hipDeviceSynchronize());
}
void LblLayer::copytoHostData(float* &imagesfloat) {
	int inputsize = inchannel*inheight*inwidth;
	imagesfloat = new float[inputsize];
	error->checkError(hipMemcpyAsync(&imagesfloat[0], ptrToOutData, sizeof(float) *inputsize, hipMemcpyDeviceToHost));
}
void LblLayer::printGrad(int dimension) {
	Layer::printGrad(dimension, "Lbl ");
}
int LblLayer::getTypeId()
{
	return LayerID::LblLayer;
}
void LblLayer::copytoDevDiff(float*labelsfloat) {
	const float scalVal = 1.0f / static_cast<float>(baSize);
	float*lbl;
	hipMalloc(&lbl, sizeof(float) * baSize);
	error->checkError(hipMemcpyAsync(lbl, labelsfloat, sizeof(float)* baSize, hipMemcpyHostToDevice));
	error->checkError(hipDeviceSynchronize());
	error->checkError(hipMemcpyAsync(&ptrToGradData[0], prevLayer->ptrToOutData, sizeof(float)* baSize* prevLayer->outchannel, hipMemcpyDeviceToDevice));
	error->checkError(hipDeviceSynchronize());
	getDiffData << <baSize, 1 >> > (lbl, ptrToGradData, prevLayer->outchannel, baSize);
	error->checkError(hipDeviceSynchronize());
	hipblasSscal(handle->cublasHandle,baSize *prevLayer->outchannel, &scalVal, ptrToGradData, 1);
	ptrToOutData = ptrToGradData;
}
void LblLayer::copytoHostLabelwComp(float* &labelsfloat) {
	int outputsize = outchannel*outheight*outwidth;
	labelsfloat = new float[outputsize*baSize];
	error->checkError(hipMemcpyAsync(labelsfloat, ptrToGradData, sizeof(float)* baSize* prevLayer->outchannel, hipMemcpyDeviceToHost));
}