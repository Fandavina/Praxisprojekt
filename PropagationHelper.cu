#include "hip/hip_runtime.h"
#include "PropagationHelper.cuh"
__global__ void Fill (const float value,float *vec, const int size)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx > size)
		return;

	vec[idx] = value;
	
}
PropagationHelper::PropagationHelper(int batchsize, int workspacesize, int workgpu)
{
	set(batchsize, workspacesize, workgpu);
}
void PropagationHelper::set(int batchsize, int workspacesize, int workgpu) {
	baSize = batchsize;
	workspaceSize = workspacesize;
	error->checkError(hipSetDevice(workgpu));
	init();
}
void PropagationHelper::init() {
	error->checkError(hipMalloc(&onevec, sizeof(float)* baSize));
	Fill << <baSize, 1 >> > (1.0f, onevec, baSize);
	error->checkError(hipDeviceSynchronize());
	error->checkError(hipMalloc(&workspace, workspaceSize));
	error->checkError(hipDeviceSynchronize());
}

void PropagationHelper::convForward(ConvLayer* conv) {
	error->checkError(hipdnnConvolutionForward(handle->cudnnHandle, &alpha,
		*conv->SrcTensor, conv->prevLayer->ptrToOutData, //x
		conv->FilterDescr, conv->ptrToDevConv,	//w
		conv->Descr, conv->AlgoFwd, workspace, workspaceSize, &beta,
		conv->DstTensor, conv->ptrToOutData));//y
	error->checkError(hipdnnAddTensor(handle->cudnnHandle, &alpha,
		conv->BiasTensorDescr, conv->ptrToDevBias,//A
		&alpha, conv->DstTensor, conv->ptrToOutData));//C
}
void PropagationHelper::poolForward( MaxPoolLayer *pool) {
	error->checkError(hipdnnPoolingForward(handle->cudnnHandle, pool->Descr, &alpha,
		*pool->SrcTensor, pool->prevLayer->ptrToOutData, &beta, //x
		pool->DstTensor, pool->ptrToOutData));//y
}
void PropagationHelper::softmaxForward(SoftMax *softmax)
{
	error->checkError(hipdnnSoftmaxForward(handle->cudnnHandle, HIPDNN_SOFTMAX_ACCURATE, HIPDNN_SOFTMAX_MODE_CHANNEL,
		&alpha, *softmax->SrcTensor, softmax->prevLayer->ptrToOutData,//x
		&beta, softmax->DstTensor, softmax->ptrToOutData));//y
}
void PropagationHelper::lrnForward(LocalResponseNormalization *lrn)
{
	error->checkError(hipdnnLRNCrossChannelForward(handle->cudnnHandle,
		lrn->Descr, HIPDNN_LRN_CROSS_CHANNEL,
		&alpha, *lrn->SrcTensor, lrn->prevLayer->ptrToOutData,//x
		&beta, lrn->DstTensor, lrn->ptrToOutData));//y
}
void PropagationHelper::activationForward(Activation *activation)
{
	error->checkError(hipdnnActivationForward(handle->cudnnHandle,
		activation->Descr, &alpha,
		*activation->SrcTensor, activation->prevLayer->ptrToOutData,//x
		&beta, activation->DstTensor, activation->ptrToOutData));//y
}
void PropagationHelper::fullyConnectedForward(FullyConnectedLayer *full)
{
	error->checkError(hipMemcpy(full->ptrToOutData, full->ptrToDevBias, full->outputsize * sizeof(float), hipMemcpyDeviceToDevice));
	gemm(HIPBLAS_OP_T, HIPBLAS_OP_N,
		full->outputsize, baSize, full->inputsize,
		alpha, beta,
		full->ptrToDevNeuron, full->inputsize,
		full->prevLayer->ptrToOutData, full->inputsize,
		full->ptrToOutData, full->outputsize);
	error->checkError(hipDeviceSynchronize());
	gemm(HIPBLAS_OP_N, HIPBLAS_OP_N,
		full->outputsize, baSize, 1,
		alpha, alpha,
		full->ptrToDevBias, full->outputsize,
		onevec, 1,
		full->ptrToOutData, full->outputsize);
	error->checkError(hipDeviceSynchronize());
}

void PropagationHelper::softmaxBackward(SoftMax *softmax) {
	error->checkError(hipdnnSoftmaxBackward(handle->cudnnHandle, HIPDNN_SOFTMAX_ACCURATE, HIPDNN_SOFTMAX_MODE_CHANNEL,
		&alpha, softmax->DstTensor, softmax->ptrToOutData,		// y
		softmax->DstTensor, softmax->nextLayer->ptrToGradData,	//dy
		&beta, *softmax->SrcTensor, softmax->ptrToGradData));	//dx
	error->checkError(hipDeviceSynchronize());
}
void PropagationHelper::fullyConnectedBackward(FullyConnectedLayer *full) {
	gemm(HIPBLAS_OP_N, HIPBLAS_OP_N,
		full->inputsize, baSize, full->outputsize, alpha, beta,
		full->ptrToDevNeuron, full->inputsize,
		full->nextLayer->ptrToGradData, full->outputsize,
		full->ptrToGradData, full->inputsize);
	error->checkError(hipDeviceSynchronize());
}
void PropagationHelper::activationBackward(Activation *activation) {
	error->checkError(hipdnnActivationBackward(handle->cudnnHandle, activation->Descr,
		&alpha, activation->DstTensor, activation->ptrToOutData,		// y
		activation->DstTensor, activation->nextLayer->ptrToGradData,	//dy
		*activation->SrcTensor, activation->prevLayer->ptrToOutData,	// x
		&beta, *activation->SrcTensor, activation->ptrToGradData));		//dx
	error->checkError(hipDeviceSynchronize());
}
void PropagationHelper::lrnBackward(LocalResponseNormalization *lrn) {
	error->checkError(hipdnnLRNCrossChannelBackward(handle->cudnnHandle, lrn->Descr, HIPDNN_LRN_CROSS_CHANNEL,
		&alpha, lrn->DstTensor, lrn->ptrToOutData,		// y
		lrn->DstTensor, lrn->nextLayer->ptrToGradData,	//dy
		*lrn->SrcTensor, lrn->prevLayer->ptrToOutData,	// x
		&beta, *lrn->SrcTensor, lrn->ptrToGradData));	//dx
	error->checkError(hipDeviceSynchronize());
}
void PropagationHelper::poolBackward(MaxPoolLayer*pool) {
	error->checkError(hipdnnPoolingBackward(handle->cudnnHandle, pool->Descr, &alpha,
		pool->DstTensor, pool->ptrToOutData,				// y
		pool->DstTensor, pool->nextLayer->ptrToGradData,	//dy
		*pool->SrcTensor, pool->prevLayer->ptrToOutData,	// x
		&beta, *pool->SrcTensor, pool->ptrToGradData));		//dx
	error->checkError(hipDeviceSynchronize());
}
void PropagationHelper::convBackward(ConvLayer*conv) {
	if (conv->dataAlgo) error->checkError(hipdnnConvolutionBackwardData(handle->cudnnHandle, &alpha,
		conv->FilterDescr, conv->ptrToDevConv,				// w
		conv->DstTensor, conv->nextLayer->ptrToGradData,	//dy
		conv->Descr, conv->AlgoDataBwd, workspace, workspaceSize,
		&beta, *conv->SrcTensor, conv->ptrToGradData));		//dx
	error->checkError(hipDeviceSynchronize());
}

void PropagationHelper::UpdateWeightsConv(float learning_rate,ConvLayer* conv,bool output)
{
	float alphal = -learning_rate;
	float* dstData;
	error->checkError(hipdnnConvolutionBackwardBias(handle->cudnnHandle, &alpha,
		conv->DstTensor, conv->nextLayer->ptrToGradData,		//dy
		&beta, conv->BiasTensorDescr, conv->ptrToGradDevBias));	//db

	error->checkError(hipdnnConvolutionBackwardFilter(handle->cudnnHandle, &alpha,
		*conv->SrcTensor, conv->prevLayer->ptrToOutData, //x
		conv->DstTensor, conv->nextLayer->ptrToGradData,//dy
		conv->Descr, conv->AlgoBwd, workspace, workspaceSize,
		&beta, conv->FilterDescr, conv->ptrToGradDevConv));//dw
	if(output){
		conv->printDevCB(5);
		conv->printGradCB(5);
		std::cout << "____________________________________________________________________" << std::endl;
	}
	Saxpy(alphal, conv->pbias.size(), conv->ptrToGradDevBias, 1, conv->ptrToDevBias, 1);

	Saxpy(alphal,conv->pconv.size(), conv->ptrToGradDevConv, 1, conv->ptrToDevConv, 1);

	error->checkError(hipDeviceSynchronize());
	if (output) {
		conv->printDevCB(5);
		std::cout << "__________________________________________________________________________________________________________________________________" << std::endl;
	}	
	error->checkError(hipDeviceSynchronize());

}
void PropagationHelper::UpdateWeightsFull(float learning_rate, FullyConnectedLayer *full,bool output) {
	float alphal = -learning_rate;
	float* dstData;
	
	if(output){
		full->printDevNB(5);
		full->printGradCB(5);
		std::cout << "____________________________________________________________________" << std::endl;
	}
	gemm(HIPBLAS_OP_N, HIPBLAS_OP_T,
		full->inputsize, full->outputsize, baSize,
		alpha, beta,
		full->prevLayer->ptrToOutData, full->inputsize,//x
		full->nextLayer->ptrToGradData, full->outputsize,//dy
		full->ptrToGradDevNeuron, full->inputsize);
	
	error->checkError(hipMalloc(&dstData, sizeof(float)* full->pneurons.size()));
	geam(HIPBLAS_OP_N, full->inputsize, full->outputsize, alphal, alpha,
		dstData, full->inputsize,
		full->ptrToGradDevNeuron, full->inputsize,
		full->ptrToDevNeuron, full->inputsize);

	gemv(full->outputsize, baSize,
		full->nextLayer->ptrToGradData, full->outputsize,//dy
		onevec, 1,
		full->ptrToGradDevBias, 1);
	error->checkError(hipDeviceSynchronize());
	error->checkError(hipMalloc(&dstData, sizeof(float)* full->pbias.size()));
	geam(HIPBLAS_OP_N, 1, full->outputsize,
		alphal, alpha,
		dstData, 1,
		full->ptrToGradDevBias, 1,
		full->ptrToDevBias, 1);

	if (output) {
		full->printDevNB(5);
		std::cout << "__________________________________________________________________________________________________________________________________" << std::endl;
	}
	error->checkError(hipFree(dstData));
	error->checkError(hipDeviceSynchronize());
}

void PropagationHelper::printDevptr(float*ptr, int size) {
	float* temp = new float[size];
	error->checkError(hipMemcpyAsync(&temp[0], ptr, size*sizeof(float), hipMemcpyDeviceToHost));
	error->checkError(hipDeviceSynchronize());
	for (int i = 0; i < size; i++) {
		std::cout << temp[i] << " ";
	}
}
void PropagationHelper::Saxpy(float alphal,size_t size,float*x, int incx, float*y,int incy) {
	error->checkError(hipblasSaxpy(handle->cublasHandle, static_cast<int>(size),	&alphal,x, incx,y, incy));
}
void PropagationHelper::gemv(int m, int n, float*A, int lda, float*x, int intcx, float*y, int intcy) {
	error->checkError(hipblasSgemv(handle->cublasHandle, HIPBLAS_OP_N, m, n, &alpha, A, lda, x, intcx, &beta, y, intcy));
	error->checkError(hipDeviceSynchronize());
}
void PropagationHelper::gemm(hipblasOperation_t transa, hipblasOperation_t transb, int m, int n, int k, float alphas, float betas, float*A, int lda, float*B, int ldb, float*C, int ldc) {
	error->checkError(hipblasSgemm(handle->cublasHandle, transa, transb, m, n, k, &alphas, A, lda, B, ldb, &betas, C, ldc));
	error->checkError(hipDeviceSynchronize());
}
void PropagationHelper::gemm(hipblasOperation_t transa, int m, int n, int k, float*A, int lda, float*B, int ldb, float*C, int ldc) {
	error->checkError(hipblasSgemm(handle->cublasHandle, transa, HIPBLAS_OP_N, m, n, k, &alpha, A, lda, B, ldb, &beta, C, ldc));
	error->checkError(hipDeviceSynchronize());
}
void PropagationHelper::geam(hipblasOperation_t transb, int m, int n, float a, float b, float*A, int lda, float*B, int ldb, float*C, int ldc) {
	error->checkError(hipblasSgeam(handle->cublasHandle, HIPBLAS_OP_N, transb, m, n, &a, A, lda, &b, B, ldb, C, ldc));
}