#include "hip/hip_runtime.h"
#include "PropagationHelper.cuh"
__global__ void Fill (const float value,float *vec, const int size)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx > size)
		return;

	vec[idx] = value;
	
}
PropagationHelper::PropagationHelper(int batchsize, int workspacesize, int workgpu)
{
	set(batchsize, workspacesize, workgpu);
}
void PropagationHelper::set(int batchsize, int workspacesize, int workgpu) {
	baSize = batchsize;
	workspaceSize = workspacesize;
	error->checkError(hipSetDevice(workgpu));
	init();
}
void PropagationHelper::init() {
	error->checkError(hipMalloc(&onevec, sizeof(float)* baSize));
	Fill << <baSize, 1 >> > (1.0f, onevec, baSize);
	error->checkError(hipDeviceSynchronize());
	error->checkError(hipMalloc(&workspace, workspaceSize));
	error->checkError(hipDeviceSynchronize());
}

void PropagationHelper::convForward(ConvLayer* conv) {
	error->checkError(hipdnnConvolutionForward(handle->cudnnHandle, &alpha,
		*conv->SrcTensor, conv->prevLayer->ptrToOutData, //x
		conv->FilterDescr, conv->ptrToDevConv,	//w
		conv->Descr, conv->AlgoFwd, workspace, workspaceSize, &beta,
		conv->DstTensor, conv->ptrToOutData));//y
	error->checkError(hipdnnAddTensor(handle->cudnnHandle, &alpha,
		conv->BiasTensorDescr, conv->ptrToDevBias,//A
		&alpha, conv->DstTensor, conv->ptrToOutData));//C
}
void PropagationHelper::poolForward( MaxPoolLayer *pool) {
	error->checkError(hipdnnPoolingForward(handle->cudnnHandle, pool->Descr, &alpha,
		*pool->SrcTensor, pool->prevLayer->ptrToOutData, &beta, //x
		pool->DstTensor, pool->ptrToOutData));//y
}
void PropagationHelper::softmaxForward(SoftMax *softmax)
{
	error->checkError(hipdnnSoftmaxForward(handle->cudnnHandle, HIPDNN_SOFTMAX_ACCURATE, HIPDNN_SOFTMAX_MODE_CHANNEL,
		&alpha, *softmax->SrcTensor, softmax->prevLayer->ptrToOutData,//x
		&beta, softmax->DstTensor, softmax->ptrToOutData));//y
}
void PropagationHelper::lrnForward(LocalResponseNormalization *lrn)
{
	error->checkError(hipdnnLRNCrossChannelForward(handle->cudnnHandle,
		lrn->Descr, HIPDNN_LRN_CROSS_CHANNEL,
		&alpha, *lrn->SrcTensor, lrn->prevLayer->ptrToOutData,//x
		&beta, lrn->DstTensor, lrn->ptrToOutData));//y
}
void PropagationHelper::activationForward(Activation *activation)
{
	error->checkError(hipdnnActivationForward(handle->cudnnHandle,
		activation->Descr, &alpha,
		*activation->SrcTensor, activation->prevLayer->ptrToOutData,//x
		&beta, activation->DstTensor, activation->ptrToOutData));//y
}
void PropagationHelper::fullyConnectedForward(FullyConnectedLayer *full)
{
	error->checkError(hipMemcpy(full->ptrToOutData, full->ptrtoDevBias, full->outchannel * sizeof(float), hipMemcpyDeviceToDevice));
	gemm(HIPBLAS_OP_T, HIPBLAS_OP_N,
		full->outchannel, baSize, full->inputsize,
		alpha, beta,
		full->ptrtoDevNeuron, full->inputsize,
		full->prevLayer->ptrToOutData, full->inputsize,
		full->ptrToOutData, full->outchannel);
	error->checkError(hipDeviceSynchronize());
	gemm(HIPBLAS_OP_N, HIPBLAS_OP_N,
		full->outchannel, baSize, 1,
		alpha, alpha,
		full->ptrtoDevBias, full->outchannel,
		onevec, 1,
		full->ptrToOutData, full->outchannel);
	error->checkError(hipDeviceSynchronize());
}

void PropagationHelper::softmaxBackward(SoftMax *softmax) {
	error->checkError(hipdnnSoftmaxBackward(handle->cudnnHandle, HIPDNN_SOFTMAX_ACCURATE, HIPDNN_SOFTMAX_MODE_CHANNEL,
		&alpha, softmax->DstTensor, softmax->ptrToOutData,		// y
		softmax->DstTensor, softmax->nextLayer->ptrToGradData,	//dy
		&beta, *softmax->SrcTensor, softmax->ptrToGradData));	//dx
	error->checkError(hipDeviceSynchronize());
}
void PropagationHelper::fullyConnectedBackward(FullyConnectedLayer *full) {
	gemm(HIPBLAS_OP_N, HIPBLAS_OP_N,
		full->inputsize, baSize, full->outchannel, alpha, beta,
		full->ptrtoDevNeuron, full->inputsize,
		full->nextLayer->ptrToGradData, full->outchannel,
		full->ptrToGradData, full->inputsize);
	error->checkError(hipDeviceSynchronize());
}
void PropagationHelper::activationBackward(Activation *activation) {
	error->checkError(hipdnnActivationBackward(handle->cudnnHandle, activation->Descr,
		&alpha, activation->DstTensor, activation->ptrToOutData,		// y
		activation->DstTensor, activation->nextLayer->ptrToGradData,	//dy
		*activation->SrcTensor, activation->prevLayer->ptrToOutData,	// x
		&beta, *activation->SrcTensor, activation->ptrToGradData));		//dx
	error->checkError(hipDeviceSynchronize());
}
void PropagationHelper::lrnBackward(LocalResponseNormalization *lrn) {
	error->checkError(hipdnnLRNCrossChannelBackward(handle->cudnnHandle, lrn->Descr, HIPDNN_LRN_CROSS_CHANNEL,
		&alpha, lrn->DstTensor, lrn->ptrToOutData,		// y
		lrn->DstTensor, lrn->nextLayer->ptrToGradData,	//dy
		*lrn->SrcTensor, lrn->prevLayer->ptrToOutData,	// x
		&beta, *lrn->SrcTensor, lrn->ptrToGradData));	//dx
	error->checkError(hipDeviceSynchronize());
}
void PropagationHelper::poolBackward(MaxPoolLayer*pool) {
	error->checkError(hipdnnPoolingBackward(handle->cudnnHandle, pool->Descr, &alpha,
		pool->DstTensor, pool->ptrToOutData,				// y
		pool->DstTensor, pool->nextLayer->ptrToGradData,	//dy
		*pool->SrcTensor, pool->prevLayer->ptrToOutData,	// x
		&beta, *pool->SrcTensor, pool->ptrToGradData));		//dx
	error->checkError(hipDeviceSynchronize());
}
void PropagationHelper::convBackward(ConvLayer*conv) {
	error->checkError(hipdnnConvolutionBackwardBias(handle->cudnnHandle, &alpha,
		conv->DstTensor, conv->nextLayer->ptrToGradData,		//dy
		&beta, conv->BiasTensorDescr, conv->ptrToGradDevBias));	//db

	error->checkError(hipdnnConvolutionBackwardFilter(handle->cudnnHandle, &alpha,
		*conv->SrcTensor, conv->prevLayer->ptrToOutData, //x
		conv->DstTensor, conv->nextLayer->ptrToGradData,//dy
		conv->Descr, conv->AlgoBwd, workspace, workspaceSize,
		&beta, conv->FilterDescr, conv->ptrToGradDevConv));//dw
	
	if (conv->dataAlgo) error->checkError(hipdnnConvolutionBackwardData(handle->cudnnHandle, &alpha,
		conv->FilterDescr, conv->ptrToDevConv,				// w
		conv->DstTensor, conv->nextLayer->ptrToGradData,	//dy
		conv->Descr, conv->AlgoDataBwd, workspace, workspaceSize,
		&beta, *conv->SrcTensor, conv->ptrToGradData));		//dx
	error->checkError(hipDeviceSynchronize());
}

void PropagationHelper::UpdateWeightsConv(float learning_rate,ConvLayer* conv,bool output)
{
	float alphal = -learning_rate;

	if(output){
		conv->printDevCB(5);
		conv->printGradCB(5);
		std::cout << "____________________________________________________________________" << std::endl;
	}
		error->checkError(hipblasSaxpy(handle->cublasHandle, static_cast<int>(conv->pconv.size()),
			&alphal, conv->ptrToGradDevConv, 1, conv->ptrToDevConv, 1));
		error->checkError(hipblasSaxpy(handle->cublasHandle, static_cast<int>(conv->pbias.size()),
			&alphal, conv->ptrToGradDevBias, 1, conv->ptrToDevBias, 1));
		error->checkError(hipDeviceSynchronize());
		if (output) {
			conv->printDevCB(5);
			std::cout << "__________________________________________________________________________________________________________________________________" << std::endl;
		}	
		error->checkError(hipDeviceSynchronize());

}
void PropagationHelper::UpdateWeightsFull(float learning_rate, FullyConnectedLayer *full,bool output) {
	float alphal = -learning_rate;
	float* dstData;
	error->checkError(hipMalloc(&dstData, sizeof(float)* full->inchannel*full->inheight*full->inwidth*full->outchannel));
	float* srcdata = full->prevLayer->ptrToOutData;
	float* diffdata = full->nextLayer->ptrToGradData;
	if(output){
		full->printDevNB(5);
		full->printGradCB(5);
		std::cout << "____________________________________________________________________" << std::endl;
	}
	gemm(HIPBLAS_OP_N, HIPBLAS_OP_T,
		full->inputsize, full->outchannel, baSize,
		alpha, beta,
		full->prevLayer->ptrToOutData, full->inputsize,
		full->nextLayer->ptrToGradData, full->outchannel,
		full->ptrToGradDevNeuron, full->inputsize);
	gemv(full->outchannel, baSize,
		full->nextLayer->ptrToGradData, full->outchannel,
		onevec, 1,
		full->ptrToGradDevBias, 1);
	error->checkError(hipblasSaxpy(handle->cublasHandle, static_cast<int>(full->pneurons.size()),
		&alpha, full->ptrToGradDevNeuron, 1, full->ptrtoDevNeuron, 1));
	error->checkError(hipblasSaxpy(handle->cublasHandle, static_cast<int>(full->pbias.size()),
		&alpha, full->ptrToGradDevBias, 1, full->ptrToGradDevBias, 1));
	//gemm(HIPBLAS_OP_N,HIPBLAS_OP_T,
	//	alpha, beta,
	//	full->inchannel, full->outchannel, baSize,
	//	srcdata, full->inchannel,
	//	diffdata, full->outchannel,
	//	dstData, full->inchannel);

	//geam(HIPBLAS_OP_N, full->inchannel, full->outchannel, alphal, alpha,
	//	dstData, full->inchannel,
	//	full->ptrtoDevNeuron, full->inchannel,
	//	full->ptrtoDevNeuron, full->inchannel);
	//
	//error->checkError(hipMalloc(&dstData, sizeof(float)* full->outchannel));
	//gemv(full->outchannel, baSize,
	//	diffdata, full->outchannel,
	//	onevec, 1,
	//	dstData, 1);
	//
	//geam(HIPBLAS_OP_N, 1, full->outchannel,
	//	alphal, alpha,
	//	dstData, 1,
	//	full->ptrtoDevBias, 1,
	//	full->ptrtoDevBias, 1);
	if (output) {
		full->printDevNB(5);
		std::cout << "__________________________________________________________________________________________________________________________________" << std::endl;
	}
	error->checkError(hipFree(dstData));
	error->checkError(hipDeviceSynchronize());
}

void PropagationHelper::printDevptr(float*ptr, int size) {
	float* temp = new float[size];
	error->checkError(hipMemcpyAsync(&temp[0], ptr, size*sizeof(float), hipMemcpyDeviceToHost));
	error->checkError(hipDeviceSynchronize());
	for (int i = 0; i < size; i++) {
		std::cout << temp[i] << " ";
	}
}

void PropagationHelper::gemv(int m, int n, float*A, int lda, float*x, int intcx, float*y, int intcy) {
	error->checkError(hipblasSgemv(handle->cublasHandle, HIPBLAS_OP_N, m, n, &alpha, A, lda, x, intcx, &beta, y, intcy));
	error->checkError(hipDeviceSynchronize());
}
void PropagationHelper::gemm(hipblasOperation_t transa, hipblasOperation_t transb, int m, int n, int k, float alphas, float betas, float*A, int lda, float*B, int ldb, float*C, int ldc) {
	error->checkError(hipblasSgemm(handle->cublasHandle, transa, transb, m, n, k, &alphas, A, lda, B, ldb, &betas, C, ldc));
	error->checkError(hipDeviceSynchronize());
}
void PropagationHelper::gemm(hipblasOperation_t transa, int m, int n, int k, float*A, int lda, float*B, int ldb, float*C, int ldc) {
	error->checkError(hipblasSgemm(handle->cublasHandle, transa, HIPBLAS_OP_N, m, n, k, &alpha, A, lda, B, ldb, &beta, C, ldc));
	error->checkError(hipDeviceSynchronize());
}
void PropagationHelper::geam(hipblasOperation_t transb, int m, int n, float a, float b, float*A, int lda, float*B, int ldb, float*C, int ldc) {
	error->checkError(hipblasSgeam(handle->cublasHandle, HIPBLAS_OP_N, transb, m, n, &a, A, lda, &b, B, ldb, C, ldc));
}