
#include "hip/hip_runtime.h"
#include ""
#include "Network.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

int main()
{

	ErrorHandler *error = &ErrorHandler::getInstance();
	Network train;
	float learningrate = 0.05;
	int trainiter = 1;
	train.train(false,false, learningrate, trainiter);
	train.test(false);

	system("Pause");
}
